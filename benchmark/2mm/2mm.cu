#include "hip/hip_runtime.h"
/**
 * 2mm.cu: This file is part of the PolyBench/GPU 1.0 test suite.
 *
 *
 * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
 * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <unistd.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

#include "../../common/polybenchUtilFuncts.h"

//define the error threshold for the results "not matching"
#define PERCENT_DIFF_ERROR_THRESHOLD 0.05

#define GPU_DEVICE 0

/* Problem size. */

# define NI 256
# define NJ 256
# define NK 256
# define NL 256

/* Thread block dimensions */
#define DIM_THREAD_BLOCK_X 32
#define DIM_THREAD_BLOCK_Y 8

#define STR_SIZE 256

/* Can switch DATA_TYPE between float and double */

typedef float DATA_TYPE;
typedef double DT;

void init_array(DATA_TYPE* A, DATA_TYPE* B, DATA_TYPE* C, DATA_TYPE* D)
{
	int i, j;
    //四个数组：从A[0]到A[NI*NK]循环赋值
    //A[i][k];B[k][j];C[l][j];D[i][l]
    //C=A*B,E=C*D，最终得到一个方阵
	for (i = 0; i < NI; i++)
	{
		for (j = 0; j < NK; j++)
		{
			A[i*NI + j] = ((DATA_TYPE) i*j) / NI;
		}
	}

	for (i = 0; i < NK; i++)
	{
		for (j = 0; j < NJ; j++)
		{
			B[i*NK + j] = ((DATA_TYPE) i*(j+1)) / NJ;
		}
	}

    for (i = 0; i < NL; i++)
	{
		for (j = 0; j < NJ; j++)
		{
			C[i*NL + j] = ((DATA_TYPE) i*(j+3)) / NL;
		}
	}

    for (i = 0; i < NI; i++)
	{
		for (j = 0; j < NL; j++)
		{
			D[i*NL + j] = ((DATA_TYPE) i*(j+2)) / NK;
		}
	}

}






//读取out.txt文件
void print(DATA_TYPE* E_outputFromGpu){
	FILE* fp;
	fp=fopen("out.txt","w");
	char str[STR_SIZE];

        if(!fp)
	{
		printf("Error writing!");
		return;
	}
	//sprintf(str,"%d",NI);
	//fputs(str,fp);
	int i,j;
    //i++与++i计算次数是没有区别的
	for (i = 0 ; i < NL ; ++i)
	{
		for (j = 0 ; j < NI ; ++j)
		{
            //sprintf函数，前两个参数固定，第一个是字符数组名，第二个是格式化
            //fputs写字符串：将字符串写入指定文件
			sprintf(str,"%f\t",E_outputFromGpu[i*NI + j]);
			fputs(str,fp);
		}
		sprintf(str,"\n");
		fputs(str,fp);
	}
	fclose(fp);
}

//对比结果
void compareResults(DATA_TYPE *E, DATA_TYPE *E_outputFromGpu)
{
	int i,j,fail;
	fail = 0;

	for (i=0; i < NL; i++)
	{
		for (j=0; j < NI; j++)
		{
			if (percentDiff(E[i*NI + j], E_outputFromGpu[i*NI + j]) > PERCENT_DIFF_ERROR_THRESHOLD)
			{
				fail++;
			}
		}
	}

	// print results
	printf("Non-Matching CPU-GPU Outputs Beyond Error Threshold of %4.2f Percent: %d\n", PERCENT_DIFF_ERROR_THRESHOLD, fail);
}

//GPU initial
void GPU_argv_init()
{
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, GPU_DEVICE);
	printf("setting device %d with name %s\n",GPU_DEVICE,deviceProp.name);
	hipSetDevice( GPU_DEVICE );
}

//两个kernel函数
__global__ void mm2_kernel1(DATA_TYPE *A, DATA_TYPE *B, DATA_TYPE *C, DT *f)
{
    //分配线程号:blockDim.x是一个threadBlock中含有thread的个数，blockIdx.x是当前是第几个threadBlock
    //threadInx.x表示当前thread在该threadblock内的index
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;

	if ((i < NI) && (j < NJ))
	{
		int k;
		for (k = 0; k < NK; k++)
		{
			C[i * NJ + j] += A[i * NK + k] * B[k * NJ + j];
		}
	}

}


__global__ void mm2_kernel2(DATA_TYPE *C, DATA_TYPE *D, DATA_TYPE *E, DT *f)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;

	if ((i < NI) && (j < NL))
	{
		int k;
		for (k = 0; k < NJ; k++)
		{
			E[i * NL + j] += C[i * NJ + k] * D[k * NL + j];
		}
	}
}


void mm2_cpu(DATA_TYPE* A, DATA_TYPE* B, DATA_TYPE* C, DATA_TYPE* D, DATA_TYPE* E)
{
	int i, j, k;

  	for (i = 0; i < NI; i++)
	{
		for (j = 0; j < NJ; j++)
		{
			C[i*NJ + j] = 0.0;
			for (k = 0; k < NK; ++k)
			{
				C[i*NJ + j] += A[i*NK + k] * B[k*NJ + j];
			}
		}
	}

	for (i = 0; i < NI; i++)
	{
		for (j = 0; j < NL; j++)
		{
			E[i*NL + j] = 0.0;
			for (k = 0; k < NJ; ++k)
			{
				E[i*NL + j] += C[i*NJ + k] * D[k*NL + j];
			}
		}
	}
}


void mm2Cuda(DATA_TYPE* A, DATA_TYPE* B, DATA_TYPE* C, DATA_TYPE* D, DATA_TYPE* E, DATA_TYPE* E_outputFromGpu, DT* f)
{
	double t_start, t_end;

	DATA_TYPE *A_gpu;
	DATA_TYPE *B_gpu;
	DATA_TYPE *C_gpu;
	DATA_TYPE *D_gpu;
	DATA_TYPE *E_gpu;
	DT *F_gpu;



	hipMalloc((void **)&A_gpu, sizeof(DATA_TYPE) * NI * NK);
	hipMalloc((void **)&B_gpu, sizeof(DATA_TYPE) * NK * NJ);
	hipMalloc((void **)&C_gpu, sizeof(DATA_TYPE) * NI * NJ);
	hipMalloc((void **)&D_gpu, sizeof(DATA_TYPE) * NJ * NL);
	hipMalloc((void **)&E_gpu, sizeof(DATA_TYPE) * NI * NL);
    hipMalloc((void **)&F_gpu, sizeof(DT) *2);


	hipMemcpy(A_gpu, A, sizeof(DATA_TYPE) * NI * NK, hipMemcpyHostToDevice);
	hipMemcpy(B_gpu, B, sizeof(DATA_TYPE) * NK * NJ, hipMemcpyHostToDevice);
	hipMemcpy(C_gpu, C, sizeof(DATA_TYPE) * NK * NJ, hipMemcpyHostToDevice);
	hipMemcpy(D_gpu, D, sizeof(DATA_TYPE) * NJ * NL, hipMemcpyHostToDevice);
	hipMemcpy(E_gpu, E, sizeof(DATA_TYPE) * NI * NL, hipMemcpyHostToDevice);
	hipMemcpy(F_gpu, f, sizeof(DT) *2, hipMemcpyHostToDevice);



	//dim3 block(32,8)：也就是说32*8个thread
	dim3 block(DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y);
	//dim3 grid1(512/32=16,512/8=64)

	dim3 grid1((size_t)ceil( ((float)NJ) / ((float)block.x) ), (size_t)ceil( ((float)NI) / ((float)block.y)) );
	dim3 grid2((size_t)ceil( ((float)NL) / ((float)block.x) ), (size_t)ceil( ((float)NI) / ((float)block.y)) );
	t_start = rtclock();
	//这是调用mm2_kernel1函数
	mm2_kernel1<<<grid1,block>>>(A_gpu, B_gpu, C_gpu, F_gpu);
	hipDeviceSynchronize();

	mm2_kernel2<<<grid2,block>>>(C_gpu, D_gpu, E_gpu, F_gpu);
	hipDeviceSynchronize();

	t_end = rtclock();
	fprintf(stdout, "GPU Runtime: %0.6lfs\n", t_end - t_start);

	hipMemcpy(E_outputFromGpu, E_gpu, sizeof(DATA_TYPE) * NI * NL, hipMemcpyDeviceToHost);

    hipMemcpy(f, F_gpu, sizeof(DT) *2, hipMemcpyDeviceToHost);

	hipFree(A_gpu);
	hipFree(B_gpu);
	hipFree(C_gpu);
	hipFree(D_gpu);
	hipFree(E_gpu);
}


int main(int argc, char** argv)
{
	double t_start, t_end;

	DATA_TYPE* C;
	DATA_TYPE* A;
	DATA_TYPE* B;
	DATA_TYPE* D;
	DATA_TYPE* E;
	DATA_TYPE* E_outputFromGpu;




	C = (DATA_TYPE*)malloc(NI*NJ*sizeof(DATA_TYPE));
	A = (DATA_TYPE*)malloc(NI*NK*sizeof(DATA_TYPE));
	B = (DATA_TYPE*)malloc(NK*NJ*sizeof(DATA_TYPE));
	D = (DATA_TYPE*)malloc(NJ*NL*sizeof(DATA_TYPE));
	E = (DATA_TYPE*)malloc(NI*NL*sizeof(DATA_TYPE));
	E_outputFromGpu = (DATA_TYPE*)malloc(NI*NL*sizeof(DATA_TYPE));

	DT* f;
    f = (DT*)malloc(2*sizeof(DT));


  	init_array(A, B, C, D);

	GPU_argv_init();

	mm2Cuda(A, B, C, D, E, E_outputFromGpu, f);

	print(E_outputFromGpu);
	printf("%x %x\n",*(int *)&(f[0]),*(int *)&(f[1]));

	//printf("%f %f\n",f[0],f[1]);
	t_start = rtclock();
	mm2_cpu(A, B, C, D, E);
	t_end = rtclock();
	fprintf(stdout, "CPU Runtime: %0.6lfs\n", t_end - t_start);
	compareResults(E, E_outputFromGpu);

	free(C);
	free(A);
	free(B);
	free(D);
	free(E);
	free(E_outputFromGpu);

  	return 0;
}
