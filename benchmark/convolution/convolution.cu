#include "hip/hip_runtime.h"
/**
 * 2DConvolution.cu: This file is part of the PolyBench/GPU 1.0 test suite.
 *
 *
 * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
 * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
 */

#include <unistd.h>
#include <stdio.h>
#include <time.h>
#include <sys/time.h>
#include <stdlib.h>
#include <stdarg.h>
#include <string.h>
#include <hip/hip_runtime.h>

#include "../../common/polybenchUtilFuncts.h"

//define the error threshold for the results "not matching"
#define PERCENT_DIFF_ERROR_THRESHOLD 0.05

#define GPU_DEVICE 0

/* Problem size */

#define NI 128
#define NJ 128

/* Thread block dimensions */
#define DIM_THREAD_BLOCK_X 32
#define DIM_THREAD_BLOCK_Y 8

/*Define size of str*/
#define STR_SIZE 256

/* Can switch DATA_TYPE between float and double */
typedef float DATA_TYPE;



void conv2D(DATA_TYPE* A, DATA_TYPE* B)
{
	int i, j;
	DATA_TYPE c11, c12, c13, c21, c22, c23, c31, c32, c33;

	c11 = +0.2;  c21 = +0.5;  c31 = -0.8;
	c12 = -0.3;  c22 = +0.6;  c32 = -0.9;
	c13 = +0.4;  c23 = +0.7;  c33 = +0.10;


	for (i = 1; i < NI - 1; ++i) // 0
	{
		for (j = 1; j < NJ - 1; ++j) // 1
		{
			B[i*NJ + j] = c11 * A[(i - 1)*NJ + (j - 1)]  +  c12 * A[(i + 0)*NJ + (j - 1)]  +  c13 * A[(i + 1)*NJ + (j - 1)]
				+ c21 * A[(i - 1)*NJ + (j + 0)]  +  c22 * A[(i + 0)*NJ + (j + 0)]  +  c23 * A[(i + 1)*NJ + (j + 0)] 
				+ c31 * A[(i - 1)*NJ + (j + 1)]  +  c32 * A[(i + 0)*NJ + (j + 1)]  +  c33 * A[(i + 1)*NJ + (j + 1)];
		}
	}
}



void init(DATA_TYPE* A)
{
	int i, j;

	for (i = 0; i < NI; ++i)
    	{
		for (j = 0; j < NJ; ++j)
		{
			A[i*NJ + j] = (float)rand()/RAND_MAX;
        	}
    	}
}

/*
void init(DATA_TYPE* A)
{
	FILE *fp;

	fp = fopen("source.txt","r");
	if(!fp){
		printf("The file was not opened\n");
		return;
	}
	
	double val=0;
	fscanf(fp,"%lf",val);
	
	int i,j;
	for(i = 0 ; i < NI ; ++i )
	    for(j = 0 ; j < NJ ; ++j)
	     {
		fscanf(fp,"%lf",A[i*NJ + j]);
             }
	fclose(fp);
}
*/
//output the result of B from GPU
void printgpub(DATA_TYPE* B_outputFromGpu){
	FILE* fp;
	fp=fopen("out.txt","w");
	char str[STR_SIZE];

        if(!fp)
	{
		printf("Error writing!");
		return;
	}
	//sprintf(str,"%d",NI);
	//fputs(str,fp);
	int i,j;

	for (i = 0 ; i < NI ; ++i)
	{
		for (j = 0 ; j < NJ ; ++j)
		{
			sprintf(str,"%e\t",B_outputFromGpu[i*NJ + j]);
			fputs(str,fp);
		}
		sprintf(str,"\n");
		fputs(str,fp);
	}
	fclose(fp);
}

/*
void printcpub(DATA_TYPE* B){
        FILE* fp;
        fp=fopen("cpu.out","w");
        char str[STR_SIZE];

        if(!fp)
        {
                printf("Error writing!");
                return;
        }
        //sprintf(str,"%d",NI);
        //fputs(str,fp);
        int i,j;

        for (i = 0 ; i < NI ; ++i)
        {
                for (j = 0 ; j < NJ ; ++j)
                {
                        sprintf(str,"%f\t",B[i*NJ + j]);
                        fputs(str,fp);
                }
                sprintf(str,"\n");
                fputs(str,fp);
        }
	fclose(fp);
}


void printa(DATA_TYPE* A){
        FILE* fp;
        fp=fopen("A.out","w");
        char str[STR_SIZE];

        if(!fp)
        {
                printf("Error writing!");
                return;
        }
        //sprintf(str,"%d",NI);
        //fputs(str,fp);
        int i,j;

        for (i = 0 ; i < NI ; ++i)
        {
                for (j = 0 ; j < NJ ; ++j)
                {
                        sprintf(str,"%f\t",A[i*NJ + j]);
                        fputs(str,fp);
                }
                sprintf(str,"\n");
                fputs(str,fp);
        }
        fclose(fp);
}

*/
void compareResults(DATA_TYPE* B, DATA_TYPE* B_outputFromGpu)
{
	int i, j, fail;
	fail = 0;
	
	// Compare a and b
	for (i=1; i < (NI-1); i++) 
	{
		for (j=1; j < (NJ-1); j++) 
		{
			if (percentDiff(B[i*NJ + j], B_outputFromGpu[i*NJ + j]) > PERCENT_DIFF_ERROR_THRESHOLD) 
			{
				fail++;
			}
		}
	}
	
	// Print results
	printf("Non-Matching CPU-GPU Outputs Beyond Error Threshold of %4.2f Percent: %d\n", PERCENT_DIFF_ERROR_THRESHOLD, fail);
	
}


void GPU_argv_init()
{
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, GPU_DEVICE);
	printf("setting device %d with name %s\n",GPU_DEVICE,deviceProp.name);
	hipSetDevice( GPU_DEVICE );
}


__global__ void Convolution2D_kernel(DATA_TYPE *A, DATA_TYPE *B)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;
    if(i==0 && j==0);
	DATA_TYPE c11, c12, c13, c21, c22, c23, c31, c32, c33;

	c11 = +0.2;  c21 = +0.5;  c31 = -0.8;
	c12 = -0.3;  c22 = +0.6;  c32 = -0.9;
	c13 = +0.4;  c23 = +0.7;  c33 = +0.10;

	if ((i < NI-1) && (j < NJ-1) && (i > 0) && (j > 0))
	{
		B[i * NJ + j] =  c11 * A[(i - 1) * NJ + (j - 1)]  + c21 * A[(i - 1) * NJ + (j + 0)] + c31 * A[(i - 1) * NJ + (j + 1)] 
			+ c12 * A[(i + 0) * NJ + (j - 1)]  + c22 * A[(i + 0) * NJ + (j + 0)] +  c32 * A[(i + 0) * NJ + (j + 1)]
			+ c13 * A[(i + 1) * NJ + (j - 1)]  + c23 * A[(i + 1) * NJ + (j + 0)] +  c33 * A[(i + 1) * NJ + (j + 1)];
	}
}


void convolution2DCuda(DATA_TYPE* A, DATA_TYPE* B, DATA_TYPE* B_outputFromGpu)
{
	double t_start, t_end;

	DATA_TYPE *A_gpu;
	DATA_TYPE *B_gpu;

	hipMalloc((void **)&A_gpu, sizeof(DATA_TYPE) * NI * NJ);
	hipMalloc((void **)&B_gpu, sizeof(DATA_TYPE) * NI * NJ);
	hipMemcpy(A_gpu, A, sizeof(DATA_TYPE) * NI * NJ, hipMemcpyHostToDevice);
	
	dim3 block(DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y);
	dim3 grid((size_t)ceil( ((float)NI) / ((float)block.x) ), (size_t)ceil( ((float)NJ) / ((float)block.y)) );
	t_start = rtclock();
	Convolution2D_kernel<<<grid,block>>>(A_gpu,B_gpu);
	hipDeviceSynchronize();
	t_end = rtclock();
	fprintf(stdout, "GPU Runtime: %0.6lfs\n", t_end - t_start);//);

	hipMemcpy(B_outputFromGpu, B_gpu, sizeof(DATA_TYPE) * NI * NJ, hipMemcpyDeviceToHost);

//	printgpub(B_outputFromGpu);
	
	hipFree(A_gpu);
	hipFree(B_gpu);
}


int main(int argc, char *argv[])
{
//	double t_start, t_end;

	DATA_TYPE* A;
	DATA_TYPE* B;  
	DATA_TYPE* B_outputFromGpu;
	
	A = (DATA_TYPE*)malloc(NI*NJ*sizeof(DATA_TYPE));
	B = (DATA_TYPE*)malloc(NI*NJ*sizeof(DATA_TYPE));
	B_outputFromGpu = (DATA_TYPE*)malloc(NI*NJ*sizeof(DATA_TYPE));

	//initialize the arrays
	init(A);
	//printa(A);
	GPU_argv_init();

	convolution2DCuda(A, B, B_outputFromGpu);
 
	printgpub(B_outputFromGpu);
	
//	t_start = rtclock();
//	conv2D(A, B);
	//printcpub(B);
//	t_end = rtclock();
//	fprintf(stdout, "CPU Runtime: %0.6lfs\n", t_end - t_start);//);
	
//	compareResults(B, B_outputFromGpu);

	free(A);
	free(B);
	free(B_outputFromGpu);
	
	return 0;
}

