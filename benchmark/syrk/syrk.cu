#include "hip/hip_runtime.h"
/**
 * syrk.cu: This file is part of the PolyBench/GPU 1.0 test suite.
 *
 *
 * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
 * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <unistd.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

#include "../../common/polybenchUtilFuncts.h"

//define the error threshold for the results "not matching"
#define PERCENT_DIFF_ERROR_THRESHOLD 0.05

#define GPU_DEVICE 0

/* Problem size */
#define N 512
#define M 512

/* Thread block dimensions */
#define DIM_THREAD_BLOCK_X 32
#define DIM_THREAD_BLOCK_Y 8

/* Declared constant values for alpha and beta (same as values in PolyBench 2.0) */
#define alpha 12435
#define beta 4546

#define STR_SIZE 256

/* Can switch DATA_TYPE between float and double */
typedef float DATA_TYPE;
typedef double DT;



void init_arrays(DATA_TYPE* A, DATA_TYPE* C)
{
	int i, j;
	
	for (i = 0; i < N; i++)
    	{
		for (j = 0; j < M; j++)
		{
			A[i*M + j] = ((DATA_TYPE) i*j) / N;
		}
		
		for (j = 0; j < N; j++)
		{
			C[i*M + j] = ((DATA_TYPE) i*j + 2) / N;
		}
	}
}


void syrk(DATA_TYPE* A, DATA_TYPE* C)
{
	int i, j, k;
	
	/*  C := alpha*A*A' + beta*C */
	for (i = 0; i < N; i++)
	{
		for (j = 0; j < N; j++)
		{
			C[i*M + j] *= beta;
		}
	}
	
	for (i = 0; i < N; i++)
	{
		for (j = 0; j < N; j++)
		{
			for (k = 0; k < M; k++)
			{
				C[i*N + j] += alpha * A[i*M + k] * A[j*M + k];
			}
		}
	}
}

void print(DATA_TYPE* C_outputFromGpu){
	FILE* fp;
	fp=fopen("out.txt","w");
	char str[STR_SIZE];

        if(!fp)
	{
		printf("Error writing!");
		return;
	}
	//sprintf(str,"%d",NI);
	//fputs(str,fp);
	int i,j;

	for (i = 1 ; i < N ; ++i)
	{
		for (j = 1 ; j < M ; ++j)
		{
			sprintf(str,"%f\t",C_outputFromGpu[i*M + j]);
			fputs(str,fp);
		}
		sprintf(str,"\n");
		fputs(str,fp);
	}
	fclose(fp);
}


void compareResults(DATA_TYPE* C, DATA_TYPE* C_outputFromGpu)
{
	int i,j,fail;
	fail = 0;

	// Compare C with D
	for (i=0; i<N; i++)
	{
		for (j=0; j<M; j++)
		{
			if (percentDiff(C[i*M + j], C_outputFromGpu[i*M + j]) > PERCENT_DIFF_ERROR_THRESHOLD)
			{
				fail++;
			}
		}
	}
	
	// print results
	printf("Non-Matching CPU-GPU Outputs Beyond Error Threshold of %4.2f Percent: %d\n", PERCENT_DIFF_ERROR_THRESHOLD, fail);
}


void GPU_argv_init()
{
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, GPU_DEVICE);
	printf("setting device %d with name %s\n",GPU_DEVICE,deviceProp.name);
	hipSetDevice( GPU_DEVICE );
	
	return;
}


__global__ void syrk_kernel(DATA_TYPE ALPHA, DATA_TYPE BETA, DATA_TYPE *a, DATA_TYPE *c, DT *f)
{
	/*  C := alpha*A*A' + beta*C */
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;

	if ((i < N) && (j < N))
	{
		c[i * N + j] *= beta;
		int k;		
		for(k=0; k< M; k++)
		{
			c[i * N + j] += alpha * a[i * M + k] * a[j * M + k];
		}
	}
}


void syrkCuda(DATA_TYPE* A, DATA_TYPE* C, DATA_TYPE* C_outputFromGpu, DT* f)
{
	double t_start, t_end;

	DATA_TYPE* A_gpu;
	DATA_TYPE* C_gpu;
	DT *F_gpu;

	hipMalloc((void **)&A_gpu, sizeof(DATA_TYPE) * N * M);
	hipMalloc((void **)&C_gpu, sizeof(DATA_TYPE) * N * N);
	hipMalloc((void **)&F_gpu, sizeof(DT) *2);
	hipMemcpy(A_gpu, A, sizeof(DATA_TYPE) * N * M, hipMemcpyHostToDevice);
	hipMemcpy(C_gpu, C, sizeof(DATA_TYPE) * N * N, hipMemcpyHostToDevice);
	hipMemcpy(F_gpu, f, sizeof(DT) *2, hipMemcpyHostToDevice);
	
	dim3 block(DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y);
	dim3 grid((size_t)(ceil(((float)N) / ((float)DIM_THREAD_BLOCK_X))), (size_t)ceil(((float)N) / ((float)DIM_THREAD_BLOCK_Y)));
	t_start = rtclock();
	syrk_kernel<<<grid,block>>>(alpha, beta, A_gpu,C_gpu, F_gpu);
	hipDeviceSynchronize();
	t_end = rtclock();
	fprintf(stdout, "GPU Runtime: %0.6lfs\n", t_end - t_start);

	hipMemcpy(C_outputFromGpu, C_gpu, sizeof(DATA_TYPE) * N * N, hipMemcpyDeviceToHost);
	hipMemcpy(f, F_gpu, sizeof(DT) *2, hipMemcpyDeviceToHost);

	hipFree(A_gpu);
	hipFree(C_gpu);
}


int main()
{
//	double t_start, t_end;

	DATA_TYPE* A;
	DATA_TYPE* C;
	DATA_TYPE* C_outputFromGpu;

	DT* f;
    f = (DT*)malloc(2*sizeof(DT));

	A = (DATA_TYPE*)malloc(N*M*sizeof(DATA_TYPE));
	C = (DATA_TYPE*)malloc(N*M*sizeof(DATA_TYPE));
	C_outputFromGpu = (DATA_TYPE*)malloc(N*M*sizeof(DATA_TYPE));

	init_arrays(A, C);
	
	GPU_argv_init();	
	syrkCuda(A, C, C_outputFromGpu,f);
	print(C_outputFromGpu);
//	t_start = rtclock();
//	syrk(A, C);
//	t_end = rtclock();
//	fprintf(stdout, "CPU Runtime: %0.6lfs\n", t_end - t_start);

//	compareResults(C, C_outputFromGpu);
    printf("%x %x\n",*(int *)&(f[0]),*(int *)&(f[1]));
	FILE* fp_reverse;
	fp_reverse=fopen("reverse.txt","a");
	//fprintf(fp_reverse,"%x %x\n",*(int *)&(f[0]),*(int *)&(f[1]));
	if(*(int *)&(f[0])<*(int *)&(f[1]))
		fprintf(fp_reverse,"%d\n",1);
	else fprintf(fp_reverse,"%d\n",0);
	fclose(fp_reverse);

	free(A);
	free(C);
	free(C_outputFromGpu);

	return 0;
}

